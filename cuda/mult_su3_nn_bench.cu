#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/resource.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "lattice.hpp"

#ifndef ITERATIONS
#  define ITERATIONS 100
#endif
#ifndef LDIM
#  define LDIM 7       // Lattice size = LDIM^4
#endif
#ifndef PRECISION
#  define PRECISION 2  // 1->single, 2->double
#endif
#ifndef VERBOSE
#  define VERBOSE 1    // valid values: 0, 1 or 2
#endif

#define CUCHECK(err, s) \
  if (err != hipSuccess) { \
        printf("%s (error code %d:%s)!\n", s, err, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
  }

//*******************  m_mat_nn.c  (in su3.a) ****************************
//  void mult_su3_nn( su3_matrix *a,*b,*c )
//  matrix multiply, no adjoints 
//  C  <-  A*B	
__global__ void k_mat_nn(
  const site* __restrict__ a,
  const su3_matrix* __restrict__ b,
  site* __restrict__ c,
  int total_sites)
{
  int mysite = blockDim.x * blockIdx.x + threadIdx.x;
#ifdef DEBUG
  printf("mysite = %d\n", mysite);
#endif

  for (int j=0; j<4; ++j) {
    for (int k=0;k<3;k++) {
      for (int l=0;l<3;l++){
        Complx cc = (0.0,0.0);
        for (int m=0;m<3;m++) {
          Complx bb = b[j].e[m][l]; __syncthreads();
          cc += a[mysite].link[j].e[k][m] * bb;
#ifdef DEBUG
          if (mysite==0 && m==2)
          printf("a[%d][%d]->e[%d][%d]=%f b[%d][%d]->e[%d][%d]=%f c[%d][%d]->e[%d][%d]=%f\n",
                  j,mysite,k,m,a[mysite].link[j].e[k][m].real(),
                  j,mysite,m,l,b[j].e[m][l].real(),
                  j,mysite,k,l,c[mysite].link[j].e[k][l].real());
#endif
        }
        c[mysite].link[j].e[k][l] = cc;
      }
    }
  }
}

int main(int argc, char *argv[])
{
  int opt;
  int threadsPerBlock=4;
  int blocksPerGrid;
  unsigned int iterations=ITERATIONS;
  unsigned int ldim=LDIM;
  unsigned int verbose=VERBOSE;

  // parse command line for parameters
  while ((opt=getopt(argc, argv, "i:n:t:v:")) != -1) {
    switch (opt) {
    case 'i':
      iterations = atoi(optarg);
      break;
    case 'n':
      ldim = atoi(optarg);
      break;
    case 't':
      threadsPerBlock = atoi(optarg);
      break;
    case 'v':
      verbose = atoi(optarg);
      break;
    default: 
      fprintf(stderr, "Usage: %s [-i iterations] [-n lattice dimension] \
[-t threads per block] [-v verbosity]\n", argv[0]);
      exit (1);
    }
  }

  // allocate and initialize the working lattices and B link matrix
  int total_sites = ldim*ldim*ldim*ldim;
  // A
  thrust::host_vector<site> a(total_sites);
  make_lattice(&a[0], ldim);
  int size_a = a.size()*sizeof(site);
  // B
  thrust::host_vector<su3_matrix> b(4);
  init_link(&b[0], Complx(1.0/3.0, 0.0));
  int size_b = b.size()*sizeof(su3_matrix);
  // C
  thrust::host_vector<site> c(total_sites);
  int size_c = c.size()*sizeof(site);

  // Device initialization
  int device;
  hipError_t cuErr;
  CUCHECK(hipGetDevice(&device), "Unable to find a device");;
  if (verbose >= 2) {
    struct hipDeviceProp_t device_prop;
    hipGetDeviceProperties(&device_prop, device);
    printf("Using device: %s\n", device_prop.name);
  }

  // Declare target storage and copy A and B
  site *d_a, *d_c;
  su3_matrix *d_b;
  cuErr = hipMalloc((void **)&d_a, total_sites*sizeof(site));
  CUCHECK(cuErr, "Unable to allocate array d_a");
  cuErr = hipMalloc((void **)&d_b, 4*sizeof(su3_matrix));
  CUCHECK(cuErr, "Unable to allocate array d_b");
  cuErr = hipMalloc((void **)&d_c, total_sites*sizeof(site));
  CUCHECK(cuErr, "Unable to allocate array d_c");
  hipMemcpy(d_a, a.data(), size_a, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b.data(), size_b, hipMemcpyHostToDevice);

  if (verbose >= 1) {
    printf("Number of sites = %d^4\n", ldim);
    printf("Executing %d iterations\n", iterations);
    printf("Threads per block set to %d\n", threadsPerBlock);
  }

  // benchmark loop
  blocksPerGrid = (total_sites + threadsPerBlock - 1)/threadsPerBlock;
  clock_t tstart = clock();
  for (int iters=0; iters<iterations; ++iters) {
      k_mat_nn<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, total_sites);
  }
  hipDeviceSynchronize();
  CUCHECK(hipGetLastError(), "k_mat_nn kernel Failed");

  double ttotal = (double)(clock()-tstart)/CLOCKS_PER_SEC;
  if (verbose >= 1)
    printf("Total execution time = %.3f secs\n", ttotal);

  // each iter of above loop is (3*3)*(12 mult + 10 add) = 108 mult + 90 add = 198 ops
  double tflop = (double)iterations * total_sites * 4.0 * 198.0;
  printf("Total GFLOP/s = %.3f\n", tflop / ttotal / 1.0e9);
  
  // copy data back from device
  hipMemcpy(c.data(), d_c, size_c, hipMemcpyDeviceToHost);

  // calculate a checksum
  double sum = 0.0;
  #pragma omp parallel for reduction(+:sum)
  for (int i=0;i<total_sites;++i) for(int j=0;j<4;++j) for(int k=0;k<3;++k) for(int l=0;l<3;++l) {
    sum += c[i].link[j].e[k][l].real();
#ifdef DEBUG
    if (i == 0)
      printf("c[%d][%d]->e[%d][%d]=%f, sum = %f\n",j,i,k,l,c[i].link[j].e[k][l].real(),sum);
#endif
  }
  sum /= (double)total_sites;

  if ( round(sum) != (4.0*sizeof(su3_matrix)/(sizeof(Complx))))
    printf("Checksum FAILED: Sum = %lf\n", sum);

  if (verbose >= 2) {
    // check memory usage
    printf("Total allocation for matrices = %.3f MiB\n", 
           ((float)sizeof(site)*(a.capacity()+c.capacity())+sizeof(su3_matrix)*b.capacity())/1048576.0);
    struct rusage usage;
    if (getrusage(RUSAGE_SELF, &usage) == 0)
      printf("Approximate memory usage = %.3f MiB\n", (float)usage.ru_maxrss/1024.0);
  }

  // Deallocate
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

}

